#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>
#include <math.h>

extern "C" {
#include "read_bmp.h"
}

#define BLACK 0
#define WHITE 255

int main(int argc, char *argv[]) {

    FILE *in_file, *out_file;
    uint8_t *input_bmp, *output_bmp;
    uint32_t width, height;
    struct timespec time_before, time_after;
    unsigned long time_dif; // in milliseconds
    uint32_t threshold, black_cell_count;
    int i, j;
    uint32_t gradient_x, gradient_y, magnitude;

    if (argc != 3) {
        fprintf(stderr, "usage: ./lab4p2_serial in_file.bmp out_file.bmp\n");
        exit(-1);
    }

    in_file = fopen(argv[1], "rb");
    out_file = fopen(argv[2], "wb");

    // Read input bmp into buffer
    input_bmp = (uint8_t *) read_bmp_file(in_file);

    // Allocate space for output image
    output_bmp = (uint8_t *) malloc(get_num_pixel());
    width = get_image_width();
    height = get_image_height();

    // Start the timer
    clock_gettime(CLOCK_REALTIME, &time_before);

    // Sobel loop
    threshold = 0;
    black_cell_count = 0;

    while (black_cell_count < (width * height * 75 / 100)) {

        black_cell_count = 0;
        threshold += 1;

        for (i = 1; i < height - 1; ++i) {
            for (j = 1; j < width -1; ++j) {
                gradient_x = input_bmp[(i - 1) * width + (j + 1)] - input_bmp[(i - 1) * width + (j - 1)] \
                            + 2 * input_bmp[i * width + (j + 1)] - 2 * input_bmp[i * width + (j - 1)] \
                            + input_bmp[(i + 1) * width + (j + 1)] - input_bmp[(i + 1) * width + (j - 1)];

                gradient_y = input_bmp[(i - 1) * width + (j - 1)] + 2 * input_bmp[(i - 1) * width + j] \
                            + input_bmp[(i - 1) * width + (j + 1)] - input_bmp[(i + 1) * width + (j - 1)] \
                            - 2 * input_bmp[(i + 1) * width + j] - input_bmp[(i + 1) * width + (j + 1)];

                magnitude = sqrt(gradient_x * gradient_x + gradient_y * gradient_y);
                if (magnitude > threshold) {
                    output_bmp[i * width + j] = WHITE;
                }
                else {
                    output_bmp[i * width + j] = BLACK;
                    ++black_cell_count;
                }
            }
        }
    }

    // End timer and print results
    clock_gettime(CLOCK_REALTIME, &time_after);
    time_dif = ((time_after.tv_sec - time_before.tv_sec) * 1000) \
               + ((time_after.tv_nsec - time_before.tv_nsec) / 1000000);

    printf("Serial time: %d (ms)\n", time_dif);
    printf("Serial threshold: %d\n", threshold);

    // Write output bmp data to file
    write_bmp_file(out_file, output_bmp);    
}
