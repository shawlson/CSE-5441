
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define MATRIX_SIZE 1024

__global__ void transpose_mult(double *matrix, double* result);

int main() {

    int i, j;
    int array_offset;
    double rand_val;
    size_t mem_size;
    double *host_matrix, *host_result;
    double *device_matrix, *device_result;
    struct timespec time_before, time_after;
    unsigned long time_dif; // in nanoseconds
    double gflops;
    
    #ifdef DEBUG
    int k;
    double *host_verify;
    int error;
    double host_entry;
    double GPU_entry;
    #endif
    
    // Allocate memory for host and device structures
    mem_size = MATRIX_SIZE * MATRIX_SIZE * sizeof(double);
    host_matrix = (double *) malloc(mem_size);
    host_result = (double *) malloc(mem_size);
    #ifdef DEBUG
    host_verify = (double *) malloc(mem_size);
    #endif
    hipMalloc((void **) &device_matrix, mem_size);
    hipMalloc((void **) &device_result, mem_size);

    /* 
     * Initialize host matrix with random values between 1.0 and
     * 2.0. Since CUDA doesn't make it easy to work with 2D arrays,
     * the matrix has been flattened to one dimension
     */
    srand(time(NULL));
    for (i = 0; i < MATRIX_SIZE; ++i) {
        for (j = 0; j < MATRIX_SIZE; ++j) {
            array_offset = i * MATRIX_SIZE;
            rand_val = 1.0 + (rand() / (RAND_MAX / (2.0 - 1.0)));
            host_matrix[array_offset + j] = rand_val;
        }
    }
    
    // Copy host matrix to GPU
    hipMemcpy(device_matrix, host_matrix, mem_size, hipMemcpyHostToDevice);

    // Start timer
    clock_gettime(CLOCK_REALTIME, &time_before);

    // Launch kernel
    dim3 blocks(MATRIX_SIZE);
    dim3 threads_per_block(MATRIX_SIZE);
    transpose_mult<<<blocks, threads_per_block>>>(device_matrix, device_result);

    // Copy result back to host and free CUDA memory
    hipMemcpy(host_result, device_result, mem_size, hipMemcpyDeviceToHost);
    hipFree(device_matrix);
    hipFree(device_result);

    // End timer and calculate GFLOPS
    clock_gettime(CLOCK_REALTIME, &time_after);
    time_dif = ((time_after.tv_sec - time_before.tv_sec) * 1000000000) + (time_after.tv_nsec - time_before.tv_nsec);
    gflops = ((double) MATRIX_SIZE * MATRIX_SIZE * MATRIX_SIZE * 2.0) / (double) time_dif;
    printf("CUDA estimated GFLOPS: %lf\n", gflops);

    #ifdef DEBUG
    // Multiple matrix by its transpose serially to verify results
    for (i = 0; i < MATRIX_SIZE; ++i) {
        for (j = 0; j < MATRIX_SIZE; ++j) {
            for (k = 0; k < MATRIX_SIZE; ++k) {
                host_verify[i * MATRIX_SIZE + j] += host_matrix[k * MATRIX_SIZE + i] \
                * host_matrix[k * MATRIX_SIZE + j];
            }
        }
    }

    error = 0;
    for (i = 0; i < MATRIX_SIZE; ++i) {
        for (j = 0; j < MATRIX_SIZE; ++j) {
            host_entry = host_verify[i * MATRIX_SIZE + j];
            GPU_entry = host_result[i * MATRIX_SIZE + j];
            if (host_entry - GPU_entry > 1.0 || host_entry - GPU_entry < -1.0) {
                error = 1;
                printf("Serial had %lf, but cuda had %lf\n");
            }
        }
    }
    
    if (!error) printf("No errors!\n");
    #endif
}

__global__ void transpose_mult(double *matrix, double *result) {

    int i, j;
    int k;
    double entry = 0.0;

    i = blockIdx.x;
    j = threadIdx.x;

    for (k = 0; k < MATRIX_SIZE; ++k) {
        // entry += matrix[k][i] * matrix[k][j];
        entry += matrix[k * MATRIX_SIZE + i] * matrix[k * MATRIX_SIZE + j];
    }

    result[i * MATRIX_SIZE + j] = entry;
}
