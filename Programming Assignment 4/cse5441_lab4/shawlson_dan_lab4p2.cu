#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>
#include <math.h>

extern "C" {
#include "read_bmp.h"
}

#define BLACK 0
#define WHITE 255

__global__ void horizontal_threads(uint8_t *input_bmp, uint8_t *output_bmp, uint32_t height, uint32_t width, uint32_t *final_thresh);
__global__ void vertical_threads(uint8_t *input_bmp, uint8_t *output_bmp, uint32_t height, uint32_t width, uint32_t *final_thresh);

int main(int argc, char *argv[]) {

    FILE *in_file, *out_file;
    uint8_t *input_bmp, *output_bmp;
    uint8_t *device_input, *device_output;
    uint32_t host_threshold, *device_threshold;
    uint32_t width, height;
    struct timespec time_before, time_after;
    unsigned long time_dif; // in milliseconds
    int num_threads;

    if (argc != 3) {
        fprintf(stderr, "usage: ./lab4p2 in_file.bmp out_file.bmp\n");
        exit(-1);
    }

    in_file = fopen(argv[1], "rb");
    out_file = fopen(argv[2], "wb");

    // Read input bmp into buffer
    input_bmp = (uint8_t *) read_bmp_file(in_file);

    // Allocate space for output image on host
    output_bmp = (uint8_t *) malloc(get_num_pixel());

    // Allocate space for input image and output image on device
    hipMalloc((void **) &device_input, get_num_pixel());
    hipMalloc((void **) &device_output, get_num_pixel());

    // Allocate space for threshold count on device, so it can be copied back
    // to the host at the end
    hipMalloc((void **) &device_threshold, sizeof(uint32_t));

    // Copy input bmp to device
    hipMemcpy(device_input, input_bmp, get_num_pixel(), hipMemcpyHostToDevice);

    width = get_image_width();
    height = get_image_height();

    // Start the timer
    clock_gettime(CLOCK_REALTIME, &time_before);

    /*
     * Start the kernel. 
     * 
     * If the picture's width is greater than or equal to its
     * height, the threads will simultaneously calculate the gradient
     * operators on the pixels of a single row.
     * 
     * If the picture's height is greater than its width,
     * the threads will simultaneously calculate the gradient
     * operators on the pixels of a single column.
     * 
     * In either case, there will be one block and as many threads as the
     * number of pixels in the dimension that's being calculated, up
     * to a maximum of 1024 threads.
     */
    if (width >= height) {
        num_threads = (width - 2 > 1024)? 1024 : width - 2;
        dim3 blocks(1);
        dim3 threads_per_block(num_threads);
        horizontal_threads<<<blocks, threads_per_block>>>(device_input, device_output, height, width, device_threshold);
    }
    else {
        num_threads = (height - 2 > 1024)? 1024 : height - 2;
        dim3 blocks(1);
        dim3 threads_per_block(num_threads);
        vertical_threads<<<blocks, threads_per_block>>>(device_input, device_output, height, width, device_threshold);
    }

    // Copy output bmp and threshold back to host and free CUDA memory
    hipMemcpy(output_bmp, device_output, get_num_pixel(), hipMemcpyDeviceToHost);
    hipMemcpy(&host_threshold, device_threshold, sizeof(uint32_t), hipMemcpyDeviceToHost);
    hipFree(device_input);
    hipFree(device_output);
    hipFree(device_threshold);

    // End timer and print results
    clock_gettime(CLOCK_REALTIME, &time_after);
    time_dif = ((time_after.tv_sec - time_before.tv_sec) * 1000) \
               + ((time_after.tv_nsec - time_before.tv_nsec) / 1000000);

    printf("CUDA time: %d (ms)\n", time_dif);
    printf("CUDA threshold: %d\n", host_threshold);

    // Write output bmp data to file
    write_bmp_file(out_file, output_bmp);    
}

__global__ void horizontal_threads(uint8_t *input_bmp, uint8_t *output_bmp, uint32_t height, uint32_t width, uint32_t *final_thresh) {

    __shared__ uint32_t black_cell_count;
    uint32_t threshold;
    int i, id;
    uint32_t gradient_x, gradient_y, magnitude;

    // We don't want any weird race conditions
    black_cell_count = 0;
    __syncthreads();

    threshold = 0;

    while (black_cell_count < (width * height * 75 / 100)) {

        // Again, no race conditions please. Let's all agree that black_cell_count
        // is 0 going forward
        black_cell_count = 0;
        __syncthreads();
        threshold += 1;

        for (i = 1; i < height - 1; ++i) {
            for (id = threadIdx.x + 1; id < width - 1; id += blockDim.x) {
                gradient_x = input_bmp[(i - 1) * width + (id + 1)] - input_bmp[(i - 1) * width + (id - 1)] \
                            + 2 * input_bmp[i * width + (id + 1)] - 2 * input_bmp[i * width + (id - 1)] \
                            + input_bmp[(i + 1) * width + (id + 1)] - input_bmp[(i + 1) * width + (id - 1)];

                gradient_y = input_bmp[(i - 1) * width + (id - 1)] + 2 * input_bmp[(i - 1) * width + id] \
                            + input_bmp[(i - 1) * width + (id + 1)] - input_bmp[(i + 1) * width + (id - 1)] \
                            - 2 * input_bmp[(i + 1) * width + id] - input_bmp[(i + 1) * width + (id + 1)];

                magnitude = sqrt((double) (gradient_x * gradient_x + gradient_y * gradient_y));

                if (magnitude > threshold) {
                    output_bmp[i * width + id] = WHITE;
                }
                else {
                    output_bmp[i * width + id] = BLACK;
                    atomicAdd(&black_cell_count, 1);
                }
            }
        }

        // Sync threads before next iteration of while loop
        __syncthreads();
    }

    // Clobber the hell out of this, it doesn't matter since all threads
    // have the same final value for threshold
    *final_thresh = threshold;
}

__global__ void vertical_threads(uint8_t *input_bmp, uint8_t *output_bmp, uint32_t height, uint32_t width, uint32_t *final_thresh) {
    
    __shared__ uint32_t black_cell_count;
    uint32_t threshold;
    int i, id;
    uint32_t gradient_x, gradient_y, magnitude;

    // We don't want any weird race conditions
    black_cell_count = 0;
    __syncthreads();

    threshold = 0;

    while (black_cell_count < (width * height * 75 / 100)) {

        // Again, no race conditions please. Let's all agree that black_cell_count
        // is 0 going forward
        black_cell_count = 0;
        __syncthreads();
        threshold += 1;

        for (i = 1; i < width - 1; ++i) {
            for (id = threadIdx.x + 1; id < height - 1; id += blockDim.x) {
                gradient_x = input_bmp[(id - 1) * width + (i + 1)] - input_bmp[(id - 1) * width + (i - 1)] \
                            + 2 * input_bmp[id * width + (i + 1)] - 2 * input_bmp[id * width + (i - 1)] \
                            + input_bmp[(id + 1) * width + (i + 1)] - input_bmp[(id + 1) * width + (i - 1)];

                gradient_y = input_bmp[(id - 1) * width + (i - 1)] + 2 * input_bmp[(id - 1) * width + i] \
                            + input_bmp[(id - 1) * width + (i + 1)] - input_bmp[(id + 1) * width + (i - 1)] \
                            - 2 * input_bmp[(id + 1) * width + i] - input_bmp[(id + 1) * width + (i + 1)];

                magnitude = sqrt((double) (gradient_x * gradient_x + gradient_y * gradient_y));

                if (magnitude > threshold) {
                    output_bmp[id * width + i] = WHITE;
                }
                else {
                    output_bmp[id * width + i] = BLACK;
                    atomicAdd(&black_cell_count, 1);
                }
            }
        }

        // Sync threads before next iteration of while loop
        __syncthreads();
    }

    // Clobber the hell out of this, it doesn't matter since all threads
    // have the same final value for threshold
    *final_thresh = threshold;
}
