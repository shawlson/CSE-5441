
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define MATRIX_SIZE 1024

int main() {

    int i, j, k;
    static double matrix[MATRIX_SIZE][MATRIX_SIZE], result[MATRIX_SIZE][MATRIX_SIZE];
    struct timespec time_before, time_after;
    unsigned long time_dif; // in nanoseconds
    double gflops;
    
    // Initialize matrix with random values between 1.0 and 2.0
    srand(time(NULL));
    for (i = 0; i < MATRIX_SIZE; ++i) {
        for (j = 0; j < MATRIX_SIZE; ++j) {
            matrix[i][j] = 1.0 + (rand() / (RAND_MAX / (2.0 - 1.0)));
        }
    }
    
    // Start timer
    clock_gettime(CLOCK_REALTIME, &time_before);

    // Multiply matrix by its transpose
    for (i = 0; i < MATRIX_SIZE; ++i) {
        for (j = 0; j < MATRIX_SIZE; ++j) {
            for (k = 0; k < MATRIX_SIZE; ++k) {
                result[i][j] += matrix[k][i] * matrix[k][j];
            }
        }
    }

    // End timer and calculate GFLOPS
    clock_gettime(CLOCK_REALTIME, &time_after);
    time_dif = ((time_after.tv_sec - time_before.tv_sec) * 1000000000) + (time_after.tv_nsec - time_before.tv_nsec);
    gflops = ((double) MATRIX_SIZE * MATRIX_SIZE * MATRIX_SIZE * 2.0) / (double) time_dif;
    printf("Serial estimated GFLOPS: %lf\n", gflops);
}
